#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_opti_simple(
	float * p, float * dp, float alpha, uint P, float div)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < P) {
		p[thx] -= alpha * dp[thx] / div;
	}
};

static __global__ void kerd_opti_simple_masque(
	float * p, float * dp, float alpha, uint P, float div, uint * masque)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < P) {
		if (masque[thx] == 0)
			p[thx] -= alpha * dp[thx] / div;
	}
};

void opti_simple(Mdl_t * mdl, float alpha, float div, uint ** masque) {
	FOR(1, c, C) {
		uint P = (mdl->ST[c-1]+1)*mdl->ST[c];
		if (masque == 0) {
			kerd_opti_simple<<<dim3(KERD(P, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha, P, div
			);
		} else {
			kerd_opti_simple_masque<<<dim3(KERD(P, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha, P, div, masque[c]
			);
		}
	};
	ATTENDRE_CUDA();
};