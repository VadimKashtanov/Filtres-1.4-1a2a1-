#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

uint poids_couche(Mdl_t * mdl, uint c) {
	return mdl->ST[c] * (mdl->ST[c-1]+1);
}

uint entrees_couche(Mdl_t * mdl, uint c) {
	return mdl->ST[c-1];
}

PAS_OPTIMISER()
Mdl_t * cree_mdl(uint * ST, uint bloques, uint f_par_bloque, uint * lignes) {
	ASSERT(ST[C-1] == 1);
	
	Mdl_t * mdl = alloc<Mdl_t>(1);

	mdl->ST = copier<uint>(ST, C);
	mdl->ST__d = cpu_vers_gpu<uint>(ST, C);

	mdl->bloques = bloques;
	mdl->f_par_bloque = f_par_bloque;

	mdl->lignes = copier<uint>(lignes, bloques);
	mdl->lignes__d = cpu_vers_gpu<uint>(lignes, bloques);

	ASSERT(ST[0] == bloques * f_par_bloque);

	//	Espace ram intel
	mdl->f = lst_rnd(ST[0] * N, -1, 1);
	mdl->dif_f = alloc<float>(ST[0]*(N-1));
	FOR(0, i, ST[0]) {
		//	Trouver min, max
		float min = mdl->f[i*N + 0];
		float max = mdl->f[i*N + 0];
		FOR(1, j, N) {
			if (mdl->f[i*N + j] < min) min = mdl->f[i*N + j];
			if (mdl->f[i*N + j] > max) max = mdl->f[i*N + j];
		}

		//	Normaliser (e-min)/(max-min)
		FOR(0, j, N) mdl->f[i*N + j] = (mdl->f[i*N + j]-min)/(max-min);

		FOR(0, j, N-1) {
			mdl->dif_f[i*(N-1) + j] = mdl->f[i*N + j+1]-mdl->f[i*N + j];
		}
	}
	mdl->f__d = cpu_vers_gpu<float>(mdl->f, ST[0] * N);
	mdl->dif_f__d = cpu_vers_gpu<float>(mdl->dif_f, ST[0] * (N-1));

	{//c = 1
		uint c = 0;
		mdl->p[c] = alloc<float>(0);
		mdl->y[c] = alloc<float>(ST[c]*PRIXS);
		mdl->locd[c] = alloc<float>(0);
		mdl->dy[c] = alloc<float>(ST[c]*PRIXS);
		mdl->dp[c] = alloc<float>(0);

		mdl->p__d[c] = alloc<float>(0);
		mdl->y__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->locd__d[c] = cudalloc<float>(0);
		mdl->dy__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->dp__d[c] = alloc<float>(0);
	}

	FOR(1, c, C) {
		mdl->p[c] = lst_rnd((ST[c-1]+1)*ST[c], -0.5, 0.5);
		mdl->y[c] = alloc<float>(ST[c]*PRIXS);
		mdl->locd[c] = alloc<float>(ST[c]*PRIXS);
		mdl->dy[c] = alloc<float>(ST[c]*PRIXS);
		mdl->dp[c] = alloc<float>((ST[c-1]+1)*ST[c]);

		mdl->p__d[c] = cpu_vers_gpu(mdl->p[c], (ST[c-1]+1)*ST[c]);
		mdl->y__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->locd__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->dy__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->dp__d[c] = cudalloc<float>((ST[c-1]+1)*ST[c]);
	}

	mdl_diff_f(mdl);

	return mdl;
};

PAS_OPTIMISER()
void mdl_verif(Mdl_t * mdl) {
	float * r = gpu_vers_cpu(mdl->dif_f__d, mdl->ST[0]*(N-1));
	FOR(0, i, (mdl->ST[0]*(N-1))) ASSERT(fabs(r[i]-mdl->dif_f[i]) < 0.01);
	free(r);
	//
	FOR(1, c, C) {
		float * r = gpu_vers_cpu(mdl->p__d[c], (mdl->ST[c-1]+1)*mdl->ST[c]);
		FOR(0, i, (mdl->ST[c-1]+1)*mdl->ST[c]) ASSERT(fabs(r[i]-mdl->p[c][i]) < 0.01);
		free(r);
	}
};

PAS_OPTIMISER()
void mdl_diff_f(Mdl_t * mdl) {
	FOR(0, i, mdl->ST[0]) {
		FOR(0, j, N-1) {
			mdl->dif_f[i*(N-1) + j] = mdl->f[i*N + j+1]-mdl->f[i*N + j];
		}
	}
	CONTROLE_CUDA(hipMemcpy(mdl->dif_f__d, mdl->dif_f, sizeof(float)*mdl->ST[0]*(N-1), hipMemcpyHostToDevice));
};

PAS_OPTIMISER()
void mdl_gpu_vers_cpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemcpy(mdl->f,     mdl->f__d,     sizeof(float)*mdl->ST[0]*N, hipMemcpyDeviceToHost));
	//
	mdl_diff_f(mdl);
	//
	CONTROLE_CUDA(hipMemcpy(mdl->y[0],  mdl->y__d[0],  sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->dy[0], mdl->dy__d[0], sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyDeviceToHost));
	FOR(1, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p[c],    mdl->p__d[c],    sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->y[c],    mdl->y__d[c],    sizeof(float)*mdl->ST[c]*PRIXS, 		   hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->locd[c], mdl->locd__d[c], sizeof(float)*mdl->ST[c]*PRIXS,  		   hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dy[c],   mdl->dy__d[c],   sizeof(float)*mdl->ST[c]*PRIXS,  		   hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dp[c],   mdl->dp__d[c],   sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyDeviceToHost));
	}
}

PAS_OPTIMISER()
void mdl_cpu_vers_gpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemcpy(mdl->f__d,     mdl->f,     sizeof(float)*mdl->ST[0]*N, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->dif_f__d,     mdl->dif_f,     sizeof(float)*mdl->ST[0]*(N-1), hipMemcpyHostToDevice));
	//
	CONTROLE_CUDA(hipMemcpy(mdl->y__d[0],  mdl->y[0],  sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->dy__d[0], mdl->dy[0], sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyHostToDevice));
	FOR(1, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],    mdl->p[c],    sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->y__d[c],    mdl->y[c],    sizeof(float)*mdl->ST[c]*PRIXS, 				   hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->locd__d[c], mdl->locd[c], sizeof(float)*mdl->ST[c]*PRIXS,  				   hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dy__d[c],   mdl->dy[c],   sizeof(float)*mdl->ST[c]*PRIXS,  				   hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dp__d[c],   mdl->dp[c],   sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void liberer_mdl(Mdl_t * mdl) {
	CONTROLE_CUDA(hipFree(mdl->ST__d));
	CONTROLE_CUDA(hipFree(mdl->lignes__d));
	CONTROLE_CUDA(hipFree(mdl->f__d));
	CONTROLE_CUDA(hipFree(mdl->dif_f__d));

	free(mdl->lignes);
	free(mdl->ST);
	free(mdl->f);
	free(mdl->dif_f);

	{
		uint c = 0;
		free(mdl->y[c]);
		free(mdl->dy[c]);

		CONTROLE_CUDA(hipFree(mdl->y__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
	}

	FOR(1, c, C) {
		free(mdl->p[c]);
		free(mdl->y[c]);
		free(mdl->locd[c]);
		free(mdl->dy[c]);
		free(mdl->dp[c]);

		CONTROLE_CUDA(hipFree(mdl->p__d[c]));
		CONTROLE_CUDA(hipFree(mdl->y__d[c]));
		CONTROLE_CUDA(hipFree(mdl->locd__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dp__d[c]));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_cpu(Mdl_t * mdl) {
	memset(mdl->dy[0], 0, sizeof(float)*mdl->ST[0]*PRIXS);
	//
	FOR(1, c, C) {
		memset(mdl->dy[c], 0, sizeof(float)*mdl->ST[c]*PRIXS);
		memset(mdl->dp[c], 0, sizeof(float)*((mdl->ST[c-1]+1)*mdl->ST[c]));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_gpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemset(mdl->dy__d[0], 0, sizeof(float)*mdl->ST[0]*PRIXS));
	//
	FOR(1, c, C) {
		CONTROLE_CUDA(hipMemset(mdl->dy__d[c], 0, sizeof(float)*mdl->ST[c]*PRIXS));
		CONTROLE_CUDA(hipMemset(mdl->dp__d[c], 0, sizeof(float)*((mdl->ST[c-1]+1)*mdl->ST[c])));
	}
};